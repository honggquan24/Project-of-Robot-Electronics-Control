/* Include files */

#include "pid_thuchanh_sfun.h"
#include "c1_pid_thuchanh.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void initialize_params_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void mdl_start_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void mdl_terminate_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance);
static void enable_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void disable_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void sf_gateway_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void ext_mode_exec_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void c1_do_animation_call_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance);
static void set_sim_state_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_st);
static void initSimStructsc1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance);
static void c1_eML_blk_kernel(SFc1_pid_thuchanhInstanceStruct *chartInstance,
  real_T c1_b_vr, real_T c1_b_vl, real_T *c1_b_v, real_T *c1_b_omega);
static real_T c1_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct *chartInstance,
  const mxArray *c1_nullptr, const char_T *c1_identifier);
static real_T c1_b_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static uint8_T c1_c_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_nullptr, const char_T *c1_identifier);
static uint8_T c1_d_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId);
static void init_dsm_address_info(SFc1_pid_thuchanhInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc1_pid_thuchanhInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c1_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c1_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  hipError_t c1_errCode;
  c1_errCode = hipGetLastError();
  if (c1_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c1_errCode), (char_T *)
                       hipGetErrorName(c1_errCode), (char_T *)
                       hipGetErrorString(c1_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c1_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c1_JITStateAnimation,
                        chartInstance->c1_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance)
{
}

static void enable_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c1_JITTransitionAnimation[0] = 0U;
  c1_eML_blk_kernel(chartInstance, *chartInstance->c1_vr, *chartInstance->c1_vl,
                    chartInstance->c1_v, chartInstance->c1_omega);
  c1_do_animation_call_c1_pid_thuchanh(chartInstance);
}

static void ext_mode_exec_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
}

static void c1_do_animation_call_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance)
{
  const mxArray *c1_b_y = NULL;
  const mxArray *c1_c_y = NULL;
  const mxArray *c1_d_y = NULL;
  const mxArray *c1_st;
  const mxArray *c1_y = NULL;
  c1_st = NULL;
  c1_st = NULL;
  c1_y = NULL;
  sf_mex_assign(&c1_y, sf_mex_createcellmatrix(3, 1), false);
  c1_b_y = NULL;
  sf_mex_assign(&c1_b_y, sf_mex_create("y", chartInstance->c1_omega, 0, 0U, 0U,
    0U, 0), false);
  sf_mex_setcell(c1_y, 0, c1_b_y);
  c1_c_y = NULL;
  sf_mex_assign(&c1_c_y, sf_mex_create("y", chartInstance->c1_v, 0, 0U, 0U, 0U,
    0), false);
  sf_mex_setcell(c1_y, 1, c1_c_y);
  c1_d_y = NULL;
  sf_mex_assign(&c1_d_y, sf_mex_create("y",
    &chartInstance->c1_is_active_c1_pid_thuchanh, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c1_y, 2, c1_d_y);
  sf_mex_assign(&c1_st, c1_y, false);
  return c1_st;
}

static void set_sim_state_c1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_st)
{
  const mxArray *c1_u;
  c1_u = sf_mex_dup(c1_st);
  *chartInstance->c1_omega = c1_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c1_u, 0)), "omega");
  *chartInstance->c1_v = c1_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c1_u, 1)), "v");
  chartInstance->c1_is_active_c1_pid_thuchanh = c1_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c1_u, 2)),
     "is_active_c1_pid_thuchanh");
  sf_mex_destroy(&c1_u);
  sf_mex_destroy(&c1_st);
}

static void initSimStructsc1_pid_thuchanh(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc1_pid_thuchanh
  (SFc1_pid_thuchanhInstanceStruct *chartInstance)
{
}

const mxArray *sf_c1_pid_thuchanh_get_eml_resolved_functions_info()
{
  const mxArray *c1_nameCaptureInfo = NULL;
  c1_nameCaptureInfo = NULL;
  sf_mex_assign(&c1_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c1_nameCaptureInfo;
}

static void c1_eML_blk_kernel(SFc1_pid_thuchanhInstanceStruct *chartInstance,
  real_T c1_b_vr, real_T c1_b_vl, real_T *c1_b_v, real_T *c1_b_omega)
{
  *c1_b_v = (c1_b_vr + c1_b_vl) / 2.0;
  *c1_b_omega = (c1_b_vr - c1_b_vl) / 0.2;
}

static real_T c1_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct *chartInstance,
  const mxArray *c1_nullptr, const char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  real_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_nullptr), &c1_thisId);
  sf_mex_destroy(&c1_nullptr);
  return c1_y;
}

static real_T c1_b_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  real_T c1_d;
  real_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_d, 1, 0, 0U, 0, 0U, 0);
  c1_y = c1_d;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static uint8_T c1_c_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_nullptr, const char_T *c1_identifier)
{
  emlrtMsgIdentifier c1_thisId;
  uint8_T c1_y;
  c1_thisId.fIdentifier = const_cast<const char_T *>(c1_identifier);
  c1_thisId.fParent = NULL;
  c1_thisId.bParentIsCell = false;
  c1_y = c1_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c1_nullptr), &c1_thisId);
  sf_mex_destroy(&c1_nullptr);
  return c1_y;
}

static uint8_T c1_d_emlrt_marshallIn(SFc1_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c1_u, const emlrtMsgIdentifier *c1_parentId)
{
  uint8_T c1_b_u;
  uint8_T c1_y;
  sf_mex_import(c1_parentId, sf_mex_dup(c1_u), &c1_b_u, 1, 3, 0U, 0, 0U, 0);
  c1_y = c1_b_u;
  sf_mex_destroy(&c1_u);
  return c1_y;
}

static void init_dsm_address_info(SFc1_pid_thuchanhInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc1_pid_thuchanhInstanceStruct
  *chartInstance)
{
  chartInstance->c1_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c1_vr = (real_T *)ssGetInputPortSignal_wrapper(chartInstance->S,
    0);
  chartInstance->c1_v = (real_T *)ssGetOutputPortSignal_wrapper(chartInstance->S,
    1);
  chartInstance->c1_vl = (real_T *)ssGetInputPortSignal_wrapper(chartInstance->S,
    1);
  chartInstance->c1_omega = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c1_pid_thuchanh_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(234149045U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2415487250U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2130554710U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2476181538U);
}

mxArray *sf_c1_pid_thuchanh_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c1_pid_thuchanh_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c1_pid_thuchanh_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c1_pid_thuchanh(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wMTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Ms4oOtnwaKfFUm/AJSfn5uanggNH1g4DZw/RM"
    "D+MCDgD0YUfzAylFHN/QoOlOmH2B9AwP2SaPEA4mcWxycml2SWpcYnG8YXZKbEl2SUJmck5mUgm"
    "QsCAGiJHpI="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c1_pid_thuchanh_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sGWqU0b5zs97DgSjNrS7ko";
}

static void sf_opaque_initialize_c1_pid_thuchanh(void *chartInstanceVar)
{
  initialize_params_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*)
    chartInstanceVar);
  initialize_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c1_pid_thuchanh(void *chartInstanceVar)
{
  enable_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c1_pid_thuchanh(void *chartInstanceVar)
{
  disable_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c1_pid_thuchanh(void *chartInstanceVar)
{
  sf_gateway_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c1_pid_thuchanh(SimStruct* S)
{
  return get_sim_state_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c1_pid_thuchanh(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c1_pid_thuchanh(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_pid_thuchanh_optimization_info();
    }

    mdl_cleanup_runtime_resources_c1_pid_thuchanh
      ((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar);
    ((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar)->
      ~SFc1_pid_thuchanhInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c1_pid_thuchanh(void *chartInstanceVar)
{
  mdl_start_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc1_pid_thuchanhInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c1_pid_thuchanh(void *chartInstanceVar)
{
  mdl_terminate_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c1_pid_thuchanh(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc1_pid_thuchanh((SFc1_pid_thuchanhInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c1_pid_thuchanh_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWN2O20QUdtLtQlEpK4EKK1Wi3HGD1BZVpRKC3eanROx2I5xtJW6WiX1iDxnPeOcn2fQNeBD",
    "uEU/AJW8A4im4RFxxxnbS4ITEk6D+ILzyOmP7m++cM+dv7NU6xx4e1/D8/T3P28Xr63jWvfy4XI",
    "xrc2d+f8f7tBj//IbnBSKECLhvBgN64bkd3CRdIkmiPPeDkwS+AiWY0VTwDh+I6ljKByCBBzhBK",
    "qR24lU0MYzyYdvwwDKrJzENYj8WhoUPcEISnnA2+Sfe1OguMjaphEC3AUIdS2GiuM1ItNoKUo8b",
    "MQRDZRJnWynQvkmtqurYME1TBq0LCDpcaYJWUGv09TXR0NAXbka2+ip/ihZJyijh1W0dE+VDit6",
    "h4TQN8f+J0Wi9irzI16ecaCEpYa2ENWIiq2K7DOU8RrdmznbWiW5C30QR5ZG1rjQJcNQf/aSCrQ",
    "YNMQJJIjjhbryB1a51kS3wzC8rYjVN4DGRhwH6gYLQLQbRgZVPcGmhh9M4YYGTPoOO6kk6wjVyz",
    "RsdG0ob5Q2T5J6kNsJmvK0RrqrakLcd8AZhTLlheyI9ghGwjL9JNNkAm/M7gJWiYU+gd9jIdYx+",
    "w+m5gQLbEDyk1b1yVEJlReIRJvwKcJrYMIAQzTwTfTbRujgySoukgaHfPDqqyLeI7XANckACqJy",
    "vJaEKUODMrxx5Q6psICEaraQzLSvPkMfgRlBPDQxvjoUcoo1dC8MzW9lIcENDGEETNGRJroXe/Z",
    "gwU1HmRGFWtu5xqjDLuvEi1sbPRuCABDGEtgZSBseYZ3GCqkusbPk8RG1HVE+aoAJJ06qRZDChY",
    "/GzVupNUjjlQy7GvC1F4hddzAq/AsCsQSTHUvYAS6mctFH4alJLOO9l2d21YbB2JpqRvvWNh8Cx",
    "GlpdbfUmAUZVi2O7iQJtg/XpU2xHuKJKYws4aWUxEGZ98IH3rA/eWdIHvzXXB+8V4+D2WUrDMx0",
    "brMI8zua5NTfP1Qr99LQPX4XzFnDeDDe9fjiHry3h9eauZb4r9b+/Xy/x1fGvVvMW7PRmiWenhN",
    "stbHX/x/1f/vzt+g/fiW8++PL7195ex19b4K9lvy0uveS2X7lWjG9Me6VZZh4tJC/77hdr/OB6y",
    "Q/sWD18cn56q3/3qbp/rxn53z6S/r2hyOf7qb5a3ksleaf3b9r+DAM3ywMy6ITFvseOicn7cTv/",
    "J3Py7q6xx5U5f8Kd3+fb4W8clPHL7HW5ZC87FglEpOS/L06Pdw7K/rizBr+Hv0b/mvw3D7bD5/z",
    "dNfLvl9ZhP9t3nBFbXeCsnMkW88um8eqKe1X4/sdVX4cqdbD+nHHec8Ztq59rfX/V31+Vj73S+3",
    "svsR6r8qhLn/ey6fWr59aHvV+MP5t9W2jElIVLdjfFY9yADJY9/Y/49x+O9nu3GLes/YoPul9/f",
    "MgJm+B2Jt8uFre70n5LnD2SQNTyPeOLqCfT67r91tVSfNvxmPJQjNVHt+/cvbNNffoLgNQKtw==",
    ""
  };

  static char newstr [1501] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c1_pid_thuchanh(SimStruct *S)
{
  const char* newstr = sf_c1_pid_thuchanh_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3608785465U));
  ssSetChecksum1(S,(2954287866U));
  ssSetChecksum2(S,(559968137U));
  ssSetChecksum3(S,(319268171U));
}

static void mdlRTW_c1_pid_thuchanh(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c1_pid_thuchanh(SimStruct *S)
{
  SFc1_pid_thuchanhInstanceStruct *chartInstance;
  chartInstance = (SFc1_pid_thuchanhInstanceStruct *)utMalloc(sizeof
    (SFc1_pid_thuchanhInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc1_pid_thuchanhInstanceStruct));
  chartInstance = new (chartInstance) SFc1_pid_thuchanhInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c1_pid_thuchanh;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c1_pid_thuchanh;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c1_pid_thuchanh;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c1_pid_thuchanh;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c1_pid_thuchanh;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c1_pid_thuchanh;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c1_pid_thuchanh;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c1_pid_thuchanh;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c1_pid_thuchanh;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c1_pid_thuchanh;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c1_pid_thuchanh;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c1_pid_thuchanh;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c1_pid_thuchanh(chartInstance);
}

void c1_pid_thuchanh_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c1_pid_thuchanh(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c1_pid_thuchanh(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c1_pid_thuchanh(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c1_pid_thuchanh_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
