/* Include files */

#include "pid_thuchanh_sfun.h"
#include "c2_pid_thuchanh.h"
#ifdef utFree
#undef utFree
#endif

#ifdef utMalloc
#undef utMalloc
#endif

#ifdef __cplusplus

extern "C" void *utMalloc(size_t size);
extern "C" void utFree(void*);

#else

extern void *utMalloc(size_t size);
extern void utFree(void*);

#endif

/* Forward Declarations */

/* Type Definitions */

/* Named Constants */
const int32_T CALL_EVENT = -1;

/* Variable Declarations */

/* Variable Definitions */
static real_T _sfTime_;

/* Function Declarations */
static void initialize_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void initialize_params_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void mdl_start_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void mdl_terminate_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void mdl_setup_runtime_resources_c2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance);
static void mdl_cleanup_runtime_resources_c2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance);
static void enable_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void disable_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void sf_gateway_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void ext_mode_exec_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void c2_do_animation_call_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static const mxArray *get_sim_state_c2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance);
static void set_sim_state_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_st);
static void initSimStructsc2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);
static void initSubchartIOPointersc2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance);
static void c2_eML_blk_kernel(SFc2_pid_thuchanhInstanceStruct *chartInstance,
  real_T c2_b_vr, real_T c2_b_vl, real_T *c2_b_v, real_T *c2_b_omega);
static real_T c2_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct *chartInstance,
  const mxArray *c2_nullptr, const char_T *c2_identifier);
static real_T c2_b_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static uint8_T c2_c_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_nullptr, const char_T *c2_identifier);
static uint8_T c2_d_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId);
static void init_dsm_address_info(SFc2_pid_thuchanhInstanceStruct *chartInstance);
static void init_simulink_io_address(SFc2_pid_thuchanhInstanceStruct
  *chartInstance);

/* Function Definitions */
static void initialize_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  emlrtLicenseCheckR2022a(chartInstance->c2_fEmlrtCtx,
    "EMLRT:runTime:MexFunctionNeedsLicense", "distrib_computing_toolbox", 2);
  sf_is_first_init_cond(chartInstance->S);
  sim_mode_is_external(chartInstance->S);
  _sfTime_ = sf_get_time(chartInstance->S);
  emlrtInitGPU(chartInstance->c2_fEmlrtCtx);
  hipGetLastError();
}

static void initialize_params_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
}

static void mdl_start_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  sim_mode_is_external(chartInstance->S);
}

static void mdl_terminate_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  hipError_t c2_errCode;
  c2_errCode = hipGetLastError();
  if (c2_errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(c2_errCode), (char_T *)
                       hipGetErrorName(c2_errCode), (char_T *)
                       hipGetErrorString(c2_errCode), (char_T *)
                       "SimGPUErrorChecks", chartInstance->c2_fEmlrtCtx);
  }
}

static void mdl_setup_runtime_resources_c2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance)
{
  sfSetAnimationVectors(chartInstance->S, chartInstance->c2_JITStateAnimation,
                        chartInstance->c2_JITTransitionAnimation);
}

static void mdl_cleanup_runtime_resources_c2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance)
{
}

static void enable_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void disable_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
}

static void sf_gateway_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  _sfTime_ = sf_get_time(chartInstance->S);
  chartInstance->c2_JITTransitionAnimation[0] = 0U;
  c2_eML_blk_kernel(chartInstance, *chartInstance->c2_vr, *chartInstance->c2_vl,
                    chartInstance->c2_v, chartInstance->c2_omega);
  c2_do_animation_call_c2_pid_thuchanh(chartInstance);
}

static void ext_mode_exec_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
}

static void c2_do_animation_call_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  sfDoAnimationWrapper(chartInstance->S, false, true);
  sfDoAnimationWrapper(chartInstance->S, false, false);
}

static const mxArray *get_sim_state_c2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance)
{
  const mxArray *c2_b_y = NULL;
  const mxArray *c2_c_y = NULL;
  const mxArray *c2_d_y = NULL;
  const mxArray *c2_st;
  const mxArray *c2_y = NULL;
  c2_st = NULL;
  c2_st = NULL;
  c2_y = NULL;
  sf_mex_assign(&c2_y, sf_mex_createcellmatrix(3, 1), false);
  c2_b_y = NULL;
  sf_mex_assign(&c2_b_y, sf_mex_create("y", chartInstance->c2_omega, 0, 0U, 0U,
    0U, 0), false);
  sf_mex_setcell(c2_y, 0, c2_b_y);
  c2_c_y = NULL;
  sf_mex_assign(&c2_c_y, sf_mex_create("y", chartInstance->c2_v, 0, 0U, 0U, 0U,
    0), false);
  sf_mex_setcell(c2_y, 1, c2_c_y);
  c2_d_y = NULL;
  sf_mex_assign(&c2_d_y, sf_mex_create("y",
    &chartInstance->c2_is_active_c2_pid_thuchanh, 3, 0U, 0U, 0U, 0), false);
  sf_mex_setcell(c2_y, 2, c2_d_y);
  sf_mex_assign(&c2_st, c2_y, false);
  return c2_st;
}

static void set_sim_state_c2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_st)
{
  const mxArray *c2_u;
  c2_u = sf_mex_dup(c2_st);
  *chartInstance->c2_omega = c2_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c2_u, 0)), "omega");
  *chartInstance->c2_v = c2_emlrt_marshallIn(chartInstance, sf_mex_dup
    (sf_mex_getcell(c2_u, 1)), "v");
  chartInstance->c2_is_active_c2_pid_thuchanh = c2_c_emlrt_marshallIn
    (chartInstance, sf_mex_dup(sf_mex_getcell(c2_u, 2)),
     "is_active_c2_pid_thuchanh");
  sf_mex_destroy(&c2_u);
  sf_mex_destroy(&c2_st);
}

static void initSimStructsc2_pid_thuchanh(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
}

static void initSubchartIOPointersc2_pid_thuchanh
  (SFc2_pid_thuchanhInstanceStruct *chartInstance)
{
}

const mxArray *sf_c2_pid_thuchanh_get_eml_resolved_functions_info()
{
  const mxArray *c2_nameCaptureInfo = NULL;
  c2_nameCaptureInfo = NULL;
  sf_mex_assign(&c2_nameCaptureInfo, sf_mex_create("nameCaptureInfo", NULL, 0,
    0U, 1U, 0U, 2, 0, 1), false);
  return c2_nameCaptureInfo;
}

static void c2_eML_blk_kernel(SFc2_pid_thuchanhInstanceStruct *chartInstance,
  real_T c2_b_vr, real_T c2_b_vl, real_T *c2_b_v, real_T *c2_b_omega)
{
  *c2_b_v = (c2_b_vr + c2_b_vl) / 2.0;
  *c2_b_omega = (c2_b_vr - c2_b_vl) / 0.2;
}

static real_T c2_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct *chartInstance,
  const mxArray *c2_nullptr, const char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  real_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_b_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_nullptr), &c2_thisId);
  sf_mex_destroy(&c2_nullptr);
  return c2_y;
}

static real_T c2_b_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  real_T c2_d;
  real_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_d, 1, 0, 0U, 0, 0U, 0);
  c2_y = c2_d;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static uint8_T c2_c_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_nullptr, const char_T *c2_identifier)
{
  emlrtMsgIdentifier c2_thisId;
  uint8_T c2_y;
  c2_thisId.fIdentifier = const_cast<const char_T *>(c2_identifier);
  c2_thisId.fParent = NULL;
  c2_thisId.bParentIsCell = false;
  c2_y = c2_d_emlrt_marshallIn(chartInstance, sf_mex_dup(c2_nullptr), &c2_thisId);
  sf_mex_destroy(&c2_nullptr);
  return c2_y;
}

static uint8_T c2_d_emlrt_marshallIn(SFc2_pid_thuchanhInstanceStruct
  *chartInstance, const mxArray *c2_u, const emlrtMsgIdentifier *c2_parentId)
{
  uint8_T c2_b_u;
  uint8_T c2_y;
  sf_mex_import(c2_parentId, sf_mex_dup(c2_u), &c2_b_u, 1, 3, 0U, 0, 0U, 0);
  c2_y = c2_b_u;
  sf_mex_destroy(&c2_u);
  return c2_y;
}

static void init_dsm_address_info(SFc2_pid_thuchanhInstanceStruct *chartInstance)
{
}

static void init_simulink_io_address(SFc2_pid_thuchanhInstanceStruct
  *chartInstance)
{
  chartInstance->c2_fEmlrtCtx = (void *)sfrtGetEmlrtCtx(chartInstance->S);
  chartInstance->c2_vr = (real_T *)ssGetInputPortSignal_wrapper(chartInstance->S,
    0);
  chartInstance->c2_v = (real_T *)ssGetOutputPortSignal_wrapper(chartInstance->S,
    1);
  chartInstance->c2_vl = (real_T *)ssGetInputPortSignal_wrapper(chartInstance->S,
    1);
  chartInstance->c2_omega = (real_T *)ssGetOutputPortSignal_wrapper
    (chartInstance->S, 2);
}

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* SFunction Glue Code */
void sf_c2_pid_thuchanh_get_check_sum(mxArray *plhs[])
{
  ((real_T *)mxGetPr((plhs[0])))[0] = (real_T)(234149045U);
  ((real_T *)mxGetPr((plhs[0])))[1] = (real_T)(2415487250U);
  ((real_T *)mxGetPr((plhs[0])))[2] = (real_T)(2130554710U);
  ((real_T *)mxGetPr((plhs[0])))[3] = (real_T)(2476181538U);
}

mxArray *sf_c2_pid_thuchanh_third_party_uses_info(void)
{
  mxArray * mxcell3p = mxCreateCellMatrix(1,0);
  return(mxcell3p);
}

mxArray *sf_c2_pid_thuchanh_jit_fallback_info(void)
{
  const char *infoFields[] = { "fallbackType", "fallbackReason",
    "hiddenFallbackType", "hiddenFallbackReason", "incompatibleSymbol" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 5, infoFields);
  mxArray *fallbackType = mxCreateString("pre");
  mxArray *fallbackReason = mxCreateString("GPUAcceleration");
  mxArray *hiddenFallbackType = mxCreateString("none");
  mxArray *hiddenFallbackReason = mxCreateString("");
  mxArray *incompatibleSymbol = mxCreateString("chartInfo");
  mxSetField(mxInfo, 0, infoFields[0], fallbackType);
  mxSetField(mxInfo, 0, infoFields[1], fallbackReason);
  mxSetField(mxInfo, 0, infoFields[2], hiddenFallbackType);
  mxSetField(mxInfo, 0, infoFields[3], hiddenFallbackReason);
  mxSetField(mxInfo, 0, infoFields[4], incompatibleSymbol);
  return mxInfo;
}

mxArray *sf_c2_pid_thuchanh_updateBuildInfo_args_info(void)
{
  mxArray *mxBIArgs = mxCreateCellMatrix(1,0);
  return mxBIArgs;
}

static const mxArray *sf_get_sim_state_info_c2_pid_thuchanh(void)
{
  const char *infoFields[] = { "chartChecksum", "varInfo" };

  mxArray *mxInfo = mxCreateStructMatrix(1, 1, 2, infoFields);
  mxArray *mxVarInfo = sf_mex_decode(
    "eNpjYPT0ZQACPiA+wMTAwAakOYCYiQECWKF8RiBmhtIQcRa4uAIQl1QWpILEi4uSPVOAdF5iLpi"
    "fWFrhmZeWDzbfggFhPhsW8xmRzOeEikPAB3vK9Ms4oOtnwaKfFUm/AJSfn5uanggNH1g4DZw/RM"
    "D+MCDgD0YUfzAylFHN/QoOlOmH2B9AwP2SaPEA4mcWxycml2SWpcYnG8UXZKbEl2SUJmck5mUgm"
    "QsCAGimHpM="
    );
  mxArray *mxChecksum = mxCreateDoubleMatrix(1, 4, mxREAL);
  sf_c2_pid_thuchanh_get_check_sum(&mxChecksum);
  mxSetField(mxInfo, 0, infoFields[0], mxChecksum);
  mxSetField(mxInfo, 0, infoFields[1], mxVarInfo);
  return mxInfo;
}

static const char* sf_get_instance_specialization(void)
{
  return "sGWqU0b5zs97DgSjNrS7ko";
}

static void sf_opaque_initialize_c2_pid_thuchanh(void *chartInstanceVar)
{
  initialize_params_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*)
    chartInstanceVar);
  initialize_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_enable_c2_pid_thuchanh(void *chartInstanceVar)
{
  enable_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_disable_c2_pid_thuchanh(void *chartInstanceVar)
{
  disable_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static void sf_opaque_gateway_c2_pid_thuchanh(void *chartInstanceVar)
{
  sf_gateway_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar);
}

static const mxArray* sf_opaque_get_sim_state_c2_pid_thuchanh(SimStruct* S)
{
  return get_sim_state_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct *)
    sf_get_chart_instance_ptr(S));     /* raw sim ctx */
}

static void sf_opaque_set_sim_state_c2_pid_thuchanh(SimStruct* S, const mxArray *
  st)
{
  set_sim_state_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*)
    sf_get_chart_instance_ptr(S), st);
}

static void sf_opaque_cleanup_runtime_resources_c2_pid_thuchanh(void
  *chartInstanceVar)
{
  if (chartInstanceVar!=NULL) {
    SimStruct *S = ((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar)->S;
    if (sim_mode_is_rtw_gen(S) || sim_mode_is_external(S)) {
      sf_clear_rtw_identifier(S);
      unload_pid_thuchanh_optimization_info();
    }

    mdl_cleanup_runtime_resources_c2_pid_thuchanh
      ((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar);
    ((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar)->
      ~SFc2_pid_thuchanhInstanceStruct();
    utFree(chartInstanceVar);
    if (ssGetUserData(S)!= NULL) {
      sf_free_ChartRunTimeInfo(S);
    }

    ssSetUserData(S,NULL);
  }
}

static void sf_opaque_mdl_start_c2_pid_thuchanh(void *chartInstanceVar)
{
  mdl_start_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*) chartInstanceVar);
  if (chartInstanceVar) {
    sf_reset_warnings_ChartRunTimeInfo(((SFc2_pid_thuchanhInstanceStruct*)
      chartInstanceVar)->S);
  }
}

static void sf_opaque_mdl_terminate_c2_pid_thuchanh(void *chartInstanceVar)
{
  mdl_terminate_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*)
    chartInstanceVar);
}

extern unsigned int sf_machine_global_initializer_called(void);
static void mdlProcessParameters_c2_pid_thuchanh(SimStruct *S)
{
  mdlProcessParamsCommon(S);
  if (sf_machine_global_initializer_called()) {
    initialize_params_c2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*)
      sf_get_chart_instance_ptr(S));
    initSubchartIOPointersc2_pid_thuchanh((SFc2_pid_thuchanhInstanceStruct*)
      sf_get_chart_instance_ptr(S));
  }
}

const char* sf_c2_pid_thuchanh_get_post_codegen_info(void)
{
  int i;
  const char* encStrCodegen [21] = {
    "eNrtWN2O20QUdtLttkWlrFTUslIlyh03SGVR1VZCsNv8lIjdboSzrcTNdmKf2NOMZ7zzk2z6Bjw",
    "I94gn4JI3APEUXCKuOOM4aXBC4klQfxBeeZ2x/c13zpnzN/YqrSMPj2t4/v6B523j9TKeVW98XM",
    "zHlZlzfH/L+zwf//yO5wUihAi4b3o9eu65HdwkbSJJojz3g5MEvgElmNFU8BbvifJYynsggQc4Q",
    "SqkduJVNDGM8n7T8MAyq6cxDWI/FoaFD3FCEh5zNvon3tToNjLWqYRANwFCHUthorjJSLTcClIP",
    "azEEfWUSZ1sp0L5JrarqyDBNUwaNcwhaXGmCVlAr9PU10VDT525Gtvoqf4IWScoo4eVtHRPlQ4r",
    "eoeEkDfH/sdFovZK8yNelnGghKWGNhNViIsti2wzlPEK3Zs521omuQ9dEEeWRta40CXDUH/2khK",
    "16NTEASSI45m68gdWucZ4t8NQvS2I1TeAJkQcB+oGC0C0G0YGVT3BpoYPTOGGBky6DlupIOsA1c",
    "s0bLRtKa+UNk4w9Sa2FzXgbA1xVtSZvM+A1wphyw3ZEeggDYBl/nWiyBnbM7wBWioYdgd5hI9cx",
    "+g2nZwZybE3wkJb3ykEBlRWJx5jwS8BpYsMAQjTzVPTpRKviyCgtkhqGfv3wsCTfPLbFNcgeCaB",
    "0vpaEKkCBM79y5A2psoGEaLSSzrQsPcM4BteCeqpneH0oZB9t7FoYXtrKRoIbGsII6qAhS3IN9O",
    "4nhJmSMicKs7J1jxOFWdaNF7E2ftYCBySIIbQ1kDI4wjyLE5RdYmXL5wFqO6B6VAcVSJqWjSSDC",
    "R2Ln7VSZ5TCCe9zMeRNKRI/72KW+BUAZg0iOZayh1hK5aiJwpeTWsJZJ8vurg2DtTPRjHStbzwC",
    "jtXQ6mqrNwkwqhoc200UaBOsT19gO8IVVRpbwFEji4Ew64P3vZd98NaCPvi9mT54Jx8He6cpDU9",
    "1bLAK8zib587MPFdL9NOTPnwZzpvDeVPc5PrxDL6ygNebuRb5rlT//n61wFfFv0rFm7PTuwWerQ",
    "JuO7fVgx93f/nztxs/fCeeffT195eur+KvzPFXst8Wl15w269cy8e3Jr3SNDMP5pKXfferFX5wo",
    "+AHdqwePT07udO9+0I9uFeP/OePpX+vL8bz/VRdLu+FgryT+7dtf4aBm+UBGbTCfN9jx8SM+3E7",
    "//0ZebdX2OPKjD/hzu/LzfC39ov4Rfa6WLCXHYsEIlLw39enx/v7RX/cWoHfwV+Df03+2/ub4cf",
    "87RXy7xbWYTfbd5wSW13gtJjJ5vPLuvHqintb+P7HlV+HMnWw+opx3ivGbaqfa31/299flo+9wv",
    "s7b7Aey/KoS5/3pun1q+fWh32Yj7+YfluoxZSFC3Y3+WPcgPQWPf2P+Pcfjva7mY8b1n75B91vP",
    "zvghI1wOzPeLua329J+S5w+kkDU4j3j66gnk+uq/dbVQnzb8ZDyUAzVJ5/u3d3bpD79BZd4Crk=",
    ""
  };

  static char newstr [1501] = "";
  newstr[0] = '\0';
  for (i = 0; i < 21; i++) {
    strcat(newstr, encStrCodegen[i]);
  }

  return newstr;
}

static void mdlSetWorkWidths_c2_pid_thuchanh(SimStruct *S)
{
  const char* newstr = sf_c2_pid_thuchanh_get_post_codegen_info();
  sf_set_work_widths(S, newstr);
  ssSetChecksum0(S,(3608785465U));
  ssSetChecksum1(S,(2954287866U));
  ssSetChecksum2(S,(559968137U));
  ssSetChecksum3(S,(319268171U));
}

static void mdlRTW_c2_pid_thuchanh(SimStruct *S)
{
  if (sim_mode_is_rtw_gen(S)) {
    ssWriteRTWStrParam(S, "StateflowChartType", "Embedded MATLAB");
  }
}

static void mdlSetupRuntimeResources_c2_pid_thuchanh(SimStruct *S)
{
  SFc2_pid_thuchanhInstanceStruct *chartInstance;
  chartInstance = (SFc2_pid_thuchanhInstanceStruct *)utMalloc(sizeof
    (SFc2_pid_thuchanhInstanceStruct));
  if (chartInstance==NULL) {
    sf_mex_error_message("Could not allocate memory for chart instance.");
  }

  memset(chartInstance, 0, sizeof(SFc2_pid_thuchanhInstanceStruct));
  chartInstance = new (chartInstance) SFc2_pid_thuchanhInstanceStruct;
  chartInstance->chartInfo.chartInstance = chartInstance;
  chartInstance->chartInfo.isEMLChart = 1;
  chartInstance->chartInfo.chartInitialized = 0;
  chartInstance->chartInfo.sFunctionGateway = sf_opaque_gateway_c2_pid_thuchanh;
  chartInstance->chartInfo.initializeChart =
    sf_opaque_initialize_c2_pid_thuchanh;
  chartInstance->chartInfo.mdlStart = sf_opaque_mdl_start_c2_pid_thuchanh;
  chartInstance->chartInfo.mdlTerminate =
    sf_opaque_mdl_terminate_c2_pid_thuchanh;
  chartInstance->chartInfo.mdlCleanupRuntimeResources =
    sf_opaque_cleanup_runtime_resources_c2_pid_thuchanh;
  chartInstance->chartInfo.enableChart = sf_opaque_enable_c2_pid_thuchanh;
  chartInstance->chartInfo.disableChart = sf_opaque_disable_c2_pid_thuchanh;
  chartInstance->chartInfo.getSimState = sf_opaque_get_sim_state_c2_pid_thuchanh;
  chartInstance->chartInfo.setSimState = sf_opaque_set_sim_state_c2_pid_thuchanh;
  chartInstance->chartInfo.getSimStateInfo =
    sf_get_sim_state_info_c2_pid_thuchanh;
  chartInstance->chartInfo.zeroCrossings = NULL;
  chartInstance->chartInfo.outputs = NULL;
  chartInstance->chartInfo.derivatives = NULL;
  chartInstance->chartInfo.mdlRTW = mdlRTW_c2_pid_thuchanh;
  chartInstance->chartInfo.mdlSetWorkWidths = mdlSetWorkWidths_c2_pid_thuchanh;
  chartInstance->chartInfo.extModeExec = NULL;
  chartInstance->chartInfo.restoreLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.restoreBeforeLastMajorStepConfiguration = NULL;
  chartInstance->chartInfo.storeCurrentConfiguration = NULL;
  chartInstance->chartInfo.callAtomicSubchartUserFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartAutoFcn = NULL;
  chartInstance->chartInfo.callAtomicSubchartEventFcn = NULL;
  chartInstance->S = S;
  chartInstance->chartInfo.dispatchToExportedFcn = NULL;
  sf_init_ChartRunTimeInfo(S, &(chartInstance->chartInfo), false, 0);
  init_dsm_address_info(chartInstance);
  init_simulink_io_address(chartInstance);
  if (!sim_mode_is_rtw_gen(S)) {
  }

  mdl_setup_runtime_resources_c2_pid_thuchanh(chartInstance);
}

void c2_pid_thuchanh_method_dispatcher(SimStruct *S, int_T method, void *data)
{
  switch (method) {
   case SS_CALL_MDL_SETUP_RUNTIME_RESOURCES:
    mdlSetupRuntimeResources_c2_pid_thuchanh(S);
    break;

   case SS_CALL_MDL_SET_WORK_WIDTHS:
    mdlSetWorkWidths_c2_pid_thuchanh(S);
    break;

   case SS_CALL_MDL_PROCESS_PARAMETERS:
    mdlProcessParameters_c2_pid_thuchanh(S);
    break;

   default:
    /* Unhandled method */
    sf_mex_error_message("Stateflow Internal Error:\n"
                         "Error calling c2_pid_thuchanh_method_dispatcher.\n"
                         "Can't handle method %d.\n", method);
    break;
  }
}
